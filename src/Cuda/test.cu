#include "hip/hip_runtime.h"
#include <Cuda/test.cuh>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void test() {
    printf("Hello World from GPU\n");
}

void testCuda() {
    // Launch kernel
    test<<<1,1>>>();
    printf("Hello world from CPU\n");
}
